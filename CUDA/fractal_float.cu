#include "hip/hip_runtime.h"
/*
Fractal code for CS 4380 / CS 5351

Copyright (c) 2019 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdio>
#include <cmath>
#include <algorithm>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "BMP43805351.h"

static const int ThreadsPerBlock = 512;

static __global__ void fractal(const int width, const int frames, unsigned char* const d_pic)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;

  if(i < frames * width * width)
  {

    const float Delta = 0.002f;
    const float xMid = 0.2315059f;
    const float yMid = 0.5214880f;

    // compute pixels of each frame
    float delta = Delta;

    const int frame = i / (width * width);

    delta = Delta * powf(0.98f, frame);

    const float xMin = xMid - delta;
    const float yMin = yMid - delta;
    const float dw = 2.0f * delta / width;

    const int row = (i / width) % width;

    const float cy = yMin + row * dw;
    
    const int col = i % width;

    const float cx = xMin + col * dw;
    float x = cx;
    float y = cy;
    float x2, y2;
    int depth = 256;
    do {
      x2 = x * x;
      y2 = y * y;
      y = 2.0f * x * y + cy;
      x = x2 - y2 + cx;
      depth--;
    } while ((depth > 0) && ((x2 + y2) < 5.0f));
    d_pic[frame * width * width + row * width + col] = (unsigned char)depth;

  }
}

static void CheckCuda()
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
    exit(-1);
  }
}

int main(int argc, char *argv[])
{
  printf("Fractal float\n");

  // check command line
  if (argc != 3) {fprintf(stderr, "USAGE: %s frame_width number_of_frames\n", argv[0]); exit(-1);}
  const int width = atoi(argv[1]);
  if (width < 10) {fprintf(stderr, "ERROR: frame_width must be at least 10\n"); exit(-1);}
  const int frames = atoi(argv[2]);
  if (frames < 1) {fprintf(stderr, "ERROR: number_of_frames must be at least 1\n"); exit(-1);}
  printf("width: %d\n", width);
  printf("frames: %d\n", frames);

  // allocate picture array
  unsigned char* pic = new unsigned char [frames * width * width];

   // allocate vectors on GPU
  unsigned char* d_pic;
  if (hipSuccess != hipMalloc((void **)&d_pic, sizeof(unsigned char) * frames * width * width)) {fprintf(stderr, "ERROR: could not allocate memory\n"); 
  exit(-1);}

  // initialize vectors on GPU
  /*
  if (hipSuccess != hipMemcpy(d_pic, pic, sizeof(unsigned char) * frames * width * width, hipMemcpyHostToDevice)) {fprintf(stderr, "ERROR: copying to device failed\n");
  exit(-1);}*/

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // execute timed code
  //fractal(width, frames, pic);
  fractal<<<((frames * width * width) + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(width, frames, d_pic);
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  const float runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0f;
  printf("compute time: %.4f s\n", runtime);

  // get result from GPU
  CheckCuda();
  if (hipSuccess != hipMemcpy(pic, d_pic, sizeof(unsigned char) * frames * width * width, hipMemcpyDeviceToHost)) {fprintf(stderr, "ERROR: copying from device failed\n"); 
  exit(-1);}

  // write result to BMP files
  if ((width <= 257) && (frames <= 60)) {
    for (int frame = 0; frame < frames; frame++) {
      BMP24 bmp(0, 0, width - 1, width - 1);
      for (int y = 0; y < width - 1; y++) {
        for (int x = 0; x < width - 1; x++) {
          const int p = pic[frame * width * width + y * width + x];
          const int e = pic[frame * width * width + y * width + (x + 1)];
          const int s = pic[frame * width * width + (y + 1) * width + x];
          const int dx = std::min(2 * std::abs(e - p), 255);
          const int dy = std::min(2 * std::abs(s - p), 255);
          bmp.dot(x, y, dx * 0x000100 + dy * 0x000001);
        }
      }
      char name[32];
      sprintf(name, "fractal%d.bmp", frame + 1000);
      bmp.save(name);
    }
  }

  // clean up
  hipFree(d_pic);
  delete [] pic;
  return 0;
}

